#include "hip/hip_runtime.h"
//nvcc -o lab5_4_2 lab5_4_2.cu
/*Author:Pedro Silva*/
/*4. Implemente um programa em CUDA que obtenha o histograma da distribuição das intensidades de uma
imagem (representada em ​ uchar​ ) com N píxeis.
4.2. Implemente uma nova versão otimizada baseada em memória partilhada..*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"


__global__ void histograma(unsigned char  *d_image, int * d_histograma, int N, int M){
	//Cada bloco vai fazer um histograma duma sub-imagem
	__shared__ int block_histograma[256];
	__shared__ unsigned char block_image[32][32];
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	//cada thread mete o seu pixel em shared mem
	if(col < N && row < M){
		block_image[threadIdx.x][threadIdx.y] = d_image[row * N + col];
		__syncthreads();
	//cada bloco tem a sua sub imagem. cada thread incrementa o histograma de acordo com o pixel em shared memory
		atomicAdd(&block_histograma[block_image[threadIdx.x][threadIdx.y]], 1);
	}
	//As primeiras 256 threads metem o histograma em shared memory para memoria global que sera transferida para host
	if(threadIdx.x + threadIdx.y * blockDim.x < 256)
		atomicAdd(&d_histograma[threadIdx.x + threadIdx.y * blockDim.x], block_histograma[threadIdx.x + threadIdx.y * blockDim.x]);
}

int main(int argc, char const *argv[]){
	printf("Exercicio 4.1, Lab 5 de CHAD. Histograma das intensidades duma imagem.\n");
	int N, M, channels, * d_histograma, * h_histograma;
	unsigned char * h_image, * d_image;
	//carregar imagem para host
	h_image = stbi_load("image_255_255_grey_gpu.png", &N, &M, &channels, 0);
	if(h_image == NULL)
		fprintf(stderr, "Erro a carregar imagem!\n");
	//alocar memoria para histograma no host
	h_histograma = (int*) malloc(256 * sizeof(int));
	//alocar memoria no device para imagem
	if(hipMalloc(&d_image, N * M * channels * sizeof(unsigned char)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para imagem.\n");
		return(-1);
	}
	if(hipMalloc(&d_histograma, 256 * sizeof(int)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para histograma.\n");
		return(-1);
	}
	//transferir imagem para device
	if(hipMemcpy(d_image, h_image, N * M * channels * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess){
		fprintf(stderr, "Erro a transferir imagem de host para device.\n");
	}
	//dimensionar grid + block
	dim3 BlockSize(32, 32, 1);
	dim3 GridSize(N / 32 + 1, M / 32 + 1, 1);
	//launch kernel
	histograma<<<GridSize, BlockSize>>>(d_image, d_histograma, N, M);
	//transferir dados de device para host
	if(hipMemcpy(h_histograma, d_histograma, 256 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
		fprintf(stderr, "Erro a transferir histograma de device para host.\n");
	}
	//imprimir histograma
	for(int i = 0; i < 255; i ++)
		printf("Intensidade %i: %i.\n", i, h_histograma[i]);
	stbi_image_free(h_image);
	free(h_histograma);
	if(hipFree(d_image) != hipSuccess)
		printf("Erro a libertar memoria no device para imagem.\n");
	if(hipFree(d_histograma) != hipSuccess)
		printf("Erro a libertar memoria no device para histograma.\n");



	return 0;

}

