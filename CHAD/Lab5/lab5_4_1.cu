#include "hip/hip_runtime.h"
//nvcc -o lab5_4_1 lab5_4_1.cu
/*Author:Pedro Silva*/
/*4. Implemente um programa em CUDA que obtenha o histograma da distribuição das intensidades de uma
imagem (representada em ​ uchar​ ) com N píxeis.
4.1. Implemente uma versão simples (sem recorrer a optimizações).*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"


__global__ void histograma(unsigned char  *d_image, int * d_histograma, int N, int M){
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	//uma operacao atomica e bloqueante, garante que nenhuma outra thread interfira, prevenindo data hazards.
	//ie: duas threads querem incrementar o valor X ao mesmo tempo. Ambas escrevem X + 1 e sera o valor final, quando deveria ser X + 2.
	if(col<N && row <M){
		atomicAdd(&d_histograma[d_image[row*N+col]], 1);
	}
}

int main(int argc, char const *argv[]){
	printf("Exercicio 4.1, Lab 5 de CHAD. Histograma das intensidades duma imagem.\n");
	int N, M, channels, * d_histograma, * h_histograma;
	unsigned char * h_image, * d_image;
	//carregar imagem para host
	h_image = stbi_load("image_255_255_grey_gpu.png", &N, &M, &channels, 0);
	if(h_image == NULL)
		fprintf(stderr, "Erro a carregar imagem!\n");
	//alocar memoria para histograma no host
	h_histograma = (int*) malloc(256 * sizeof(int));
	//alocar memoria no device para imagem
	if(hipMalloc(&d_image, N * M * channels * sizeof(unsigned char)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para imagem.\n");
		return(-1);
	}
	if(hipMalloc(&d_histograma, 256 * sizeof(int)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para histograma.\n");
		return(-1);
	}
	//transferir imagem para device
	if(hipMemcpy(d_image, h_image, N * M * channels * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess){
		fprintf(stderr, "Erro a transferir imagem de host para device.\n");
	}
	//dimensionar grid + block
	dim3 BlockSize(32, 32, 1);
	dim3 GridSize(N / 32 + 1, M / 32 + 1, 1);
	//launch kernel
	histograma<<<GridSize, BlockSize>>>(d_image, d_histograma, N, M);
	//transferir dados de device para host
	if(hipMemcpy(h_histograma, d_histograma, 256 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
		fprintf(stderr, "Erro a transferir histograma de device para host.\n");
	}
	//imprimir histograma
	for(int i = 0; i < 255; i ++)
		printf("Intensidade %i: %i.\n", i, h_histograma[i]);
	stbi_image_free(h_image);
	free(h_histograma);
	if(hipFree(d_image) != hipSuccess)
		printf("Erro a libertar memoria no device para imagem.\n");
	if(hipFree(d_histograma) != hipSuccess)
		printf("Erro a libertar memoria no device para histograma.\n");



	return 0;

}

