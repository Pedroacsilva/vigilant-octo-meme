#include "hip/hip_runtime.h"
//nvcc -o lab5_2 lab5_2.cu
/*Author:
Pedro Silva
*/
/*2. Implemente um programa em CUDA que calcule a soma de todos os elementos de um vetor de
tamanho N. Teste para vários valores de N.*/
/*2.1. Implemente uma versão simples (sem recorrer a optimizações).*/

__global__ void vectorsum2_1(int * device_buffer){
	//THREAD ID
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	/*Temos N elementos no vector. Vamos partir a soma de todos os elementos como a soma de um elemento com o elemento à sua direita.
	Repetir até so termos um elemento (a cada iteração, temos metade dos elementos).*/
	//Assumir que só lançamos um bloco de threads (blockIdx.x = 1 para todas as threads.)
	int num_of_threads = blockDim.x;			//Se só lançamos um bloco...
	int distance = 1;			//Distancia entre elemtnos a somar

	/*Este ciclo executa enquanto tivermos mais que uma thread e so se a thread ainda estiver no "scope" da soma.*/
	while(num_of_threads > 0 && index < num_of_threads){
		int primeiro = index * distance * 2;		//na primeira iteracao: 1a thread, index 0, 2a thread, index 2, 3a thread, index 4
		int segundo = primeiro + distance;			//na primeira iteracao: 1a thread, index 1, 2a thread, index 3, 3a thread, index 5
		device_buffer[primeiro] = device_buffer[primeiro] + device_buffer[segundo];

		//passou uma iteracao: duplicar a distancia entre elementos a somar e dividir por 2 o numero de threads activas
		distance = distance * 2;
		num_of_threads = num_of_threads / 2;

	}
}		

int main(){
	//32768 = 2¹⁵;
	for(int N = 8; N <= 32768; N = N*2){
		int *device_buffer = NULL;
		int err = hipMalloc(&device_buffer, sizeof(int) * N);
		if(err != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		vectorsum2_1<<< 1, N/2 >>>(device_buffer);
		int result;
		/*Vamos buscar o resultado da soma ao primeiro elemento do device_buffer*/
		hipMemcpy(&result, device_buffer, sizeof(int));
		printf("Resultado da soma de um vector de %i elementos: %i.\n", N, result);
	}
	
	return 0;
}


