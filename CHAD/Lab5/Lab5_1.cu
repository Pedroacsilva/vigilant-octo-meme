#include "hip/hip_runtime.h"
/*Author:
Pedro Silva
*/

// Analise o seguinte kernel, ​ colorToGreyScaleConversion()​ , que produz a conversão de cor
// para escala de cinzentos de uma imagem com mapeamento de dados em threads 2D
// Device code
// We have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ void colorToGreyScaleConvertion(unsigned char * grayImage, unsigned char
	*rgbImage, int width, int height)
{
	int Col=threadIdx.x+(blockIdx.x*blockDim.x);
	int Row=threadIdx.y+(blockIdx.y*blockDim.y);
	if (Col<width && Row<height){
// get 1D coordinate for the grayscale image
		int greyOffset=Row*width + Col;
// one can think of the RGB image having
// CHANNEL times columns of the gray scale image
		int rgbOffset=greyOffset*CHANNELS;
		unsigned char r=rgbImage[rgbOffset];
// red value for pixel
		unsigned char g=rgbImage[rgbOffset+1];
// green value for pixel
		unsigned char b=rgbImage[rgbOffset+2];
// blue value for pixel
// perform the rescaling and store it
// We multiply by floating point constants
		grayImage[grayOffset]=0.21f*r + 0.71f*g + 0.07f*b;
	}
}