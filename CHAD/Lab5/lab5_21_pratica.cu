//nvcc -o lab5_21 lab5_21.cu
/*Author:Pedro Silva*/
/*2. Implemente um programa em CUDA que calcule a soma de todos os elementos de um vetor de
tamanho N. Teste para vários valores de N.*/
/*2.1. Implemente uma versão simples (sem recorrer a optimizações).*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
__global__ void vectorsum2_1(int * device_buffer, int N){
	//THREAD ID
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	/*Temos N elementos no vector. Vamos partir a soma de todos os elementos como a soma de um elemento com o elemento à sua direita.
	Repetir até so termos um elemento (a cada iteração, temos metade dos elementos).*/
	//Assumir que só lançamos um bloco de threads (blockIdx.x = 1 para todas as threads.)
	////int num_of_threads = blockDim.x;
	int distance = blockDim.x;			//Distancia inicial entre elementos a somar
    int num_of_blocks = N / blockDim.x + 1;
	/*Este ciclo executa enquanto tivermos mais que uma thread e so se a thread ainda estiver no "scope" da soma.*/
	while(num_of_blocks > 1 && blockIdx.x < num_of_blocks && index < N){
        
		int primeiro = index * distance * 2;		//na primeira iteracao: 1a thread, index 0, 2a thread, index 2, 3a thread, index 4
		int segundo = primeiro + distance;			//na primeira iteracao: 1a thread, index 1, 2a thread, index 3, 3a thread, index 5
		printf("DEVICE: Thread %i. A somar %i + %i\n", index, device_buffer[primeiro], device_buffer[segundo]);
		device_buffer[primeiro] = device_buffer[primeiro] + device_buffer[segundo];
		//passou uma iteracao: duplicar a distancia entre elementos a somar e dividir por 2 o numero de threads activas
        distance *= 2;
        num_of_blocks--;
	}
	int num_of_threads = blockDim.x/2;
	if(num_of_blocks == 1 && num_of_threads > 1){
        int primeiro = index * 2;
        int segundo = primeiro + 1;
        device_buffer[primeiro] = primeiro + segundo;
        num_of_threads /=2;
    }
}		

int main(){
    struct timespec start_device, end_device, start_host, end_host;
    double initialTime, finalTime;
    int result;

    for(int N = 8; N <= 1024; N = N*2){
        printf("N = %i.\n", N);
        int *device_buffer = NULL;
        int *host_buffer = NULL;
		int err = hipMalloc(&device_buffer, sizeof(int) * N);
		if(err != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		//Inicializar vector N:
		
        host_buffer = (int*)malloc(N * sizeof(int));
        for(int i = 0; i < N; i++)
            host_buffer[i] = i;
        //DEVICE
        //enviar dados para device
        hipMemcpy(device_buffer, host_buffer, N * sizeof(int), hipMemcpyHostToDevice);
        //comecar computacao
        clock_gettime(CLOCK_MONOTONIC, &start_device);
		vectorsum2_1<<< N/256 + 1, 256>>>(device_buffer, N);
        clock_gettime(CLOCK_MONOTONIC, &end_device);
        //cronometrar
        initialTime = (start_device.tv_sec*1e3) + (start_device.tv_nsec*1e-6);
        finalTime = (end_device.tv_sec*1e3) + (end_device.tv_nsec*1e-6);
		/*Vamos buscar o resultado da soma ao primeiro elemento do device_buffer*/
		hipMemcpy(&result, device_buffer, sizeof(int), hipMemcpyDeviceToHost);
		printf("DEVICE: Resultado da soma de um vector de %i elementos: %i.\n", N, result);
        printf("DEVICE: Tempo de execução (device): \t%fms.\n", (finalTime - initialTime));
        //HOST
        result = 0;
        clock_gettime(CLOCK_MONOTONIC, &start_host);
        for(int i = 0; i < N; i++)
            result += host_buffer[i];
        clock_gettime(CLOCK_MONOTONIC, &end_host);
        initialTime = (start_host.tv_sec*1e3) + (start_host.tv_nsec*1e-6);
        finalTime = (end_host.tv_sec*1e3) + (end_host.tv_nsec*1e-6);
        printf("HOST: Resultado da soma de um vector de %i elementos: %i.\n", N, result);
        printf("HOST: Tempo de execução (device): \t%fms.\n", (finalTime - initialTime));
        hipFree(device_buffer);
        free(host_buffer);
        return 0;       //TEMPORARIO. So quero testar para N = 8
	}
	
	return 0;
}


