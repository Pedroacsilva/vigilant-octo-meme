#include "hip/hip_runtime.h"
//nvcc -o lab5_1_cpu lab5_1_cpu.cu
//Implementacao em CPU para a conversao em tons de cinza duma imagem RGB
//Autor: Pedro Silva

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
/*stb -> biblioteca para abrir e ler imagens: https://github.com/nothings/stb*/

#define CHANNELS 4		//(RGBA)


// Device code
// We have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
//__global__
void colorToGreyScaleConvertion(unsigned char* grayImage, unsigned char
	* rgbImage, int width, int height)
{
	for(int col=0; col<width; col++)
    {
        for(int row=0; row<height; row++)
        {
            // get 1D coordinate for the grayscale image
            int grayOffset = row * width + col;
            // one can think of the RGB image having
            // CHANNEL times columns of the gray scale image
            int rgbOffset = grayOffset * CHANNELS;
            unsigned char r = rgbImage[rgbOffset];
            // red value for pixel
            unsigned char g = rgbImage[rgbOffset + 1];
            // green value for pixel
            unsigned char b = rgbImage[rgbOffset + 2];
            // blue value for pixel
            // perform the rescaling and store it
            // We multiply by floating point constants
            grayImage[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
        }
	}
}


int main(int argc, char const *argv[])
{
	printf("Exercicio 1, Lab 5 de CHAD. Leitura e grayscaling de imagens com varias dimensoes (CPU).\n");


	int N, M, channels;
	unsigned char * h_i_rgb, * h_i_gs;
	struct timespec start, end;
	double startTime, endTime;


	for(int i = 0; i < 5; i++){
		printf("%i-ésima iteração.\n", i);
		//Definir as dimensoes da imagem, N e M, de acordo com a iteracao, i, do ciclo e load da imagem
		//unsigned char *data = stbi_load(filename, &x, &y, &n, 0);
		// Standard parameters:
//    int *x                 -- outputs image width in pixels
//    int *y                 -- outputs image height in pixels
//    int *channels_in_file  -- outputs # of image components in image file
//    int desired_channels   -- if non-zero, # of image components requested in result
		//int i = 1;
		switch(i){
			case 0:
			h_i_rgb = stbi_load("image_255_255.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 1:
			h_i_rgb = stbi_load("image_800_600.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 2:
			h_i_rgb = stbi_load("image_1920_1080.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 3:
			h_i_rgb = stbi_load("image_3840_2160.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 4:
			h_i_rgb = stbi_load("image_7680_4320.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			default:
			fprintf(stderr, "Erro no switch para inicializar dimensoes da imagem.\n");
			return(-1);
		}
		//Alocar memoria para imagem GS
		h_i_gs = (unsigned char*)malloc(N * M * 1 * sizeof(unsigned char));		//So temos um canal de cinzentos
		//comecar conversao e cronometrar
		clock_gettime(CLOCK_MONOTONIC, &start);
		colorToGreyScaleConvertion(h_i_gs, h_i_rgb, N, M);
		//Gravar imagem em ficheiros
		//int stbi_write_png(char const *filename, int w, int h, int comp, const void *data, int stride_in_bytes);
		switch(i){
			case 0:
			if(stbi_write_png("image_255_255_grey_cpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 1:
			if(stbi_write_png("image_800_600_grey_cpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 2:
			if(stbi_write_png("image_1920_1080_grey_cpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 3:
			if(stbi_write_png("image_3840_2160_grey_cpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 4:
			if(stbi_write_png("image_7680_4320_grey_cpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			default:
			fprintf(stderr, "Erro no switch de imprimir imagem.\n");
			return(-1);
		}

		//Memory cleanup
		stbi_image_free(h_i_rgb);
		free(h_i_gs);
		clock_gettime(CLOCK_MONOTONIC, &end);
		startTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
		endTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
		printf("Tempo de execução do CPU: %fms.\n", endTime - startTime);
	}
	return 0;
}