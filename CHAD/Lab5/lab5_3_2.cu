//nvcc -o lab5_3_2 lab5_3_2.cu
/*Author:
Pedro Silva
*/
/*3. Implemente um programa em CUDA que devolva a transposta de uma matriz*/
/*3.2. Implemente uma nova versão otimizada baseada em memória partilhada (​ shared memory ) ​ .*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ void transposta(int *d_matrix, int *d_out, int N){
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col_t = threadIdx.y + blockIdx.y * blockDim.y;
	int row_t = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int shared_mem[32][32];
	if(col<N && row <N)
	{
		shared_mem[threadIdx.y][threadIdx.x] = d_matrix[row*N+col];
		d_out[row_t*N+col_t] = shared_mem[threadIdx.y][threadIdx.x];
	}
}

int main(int argc, char const *argv[])
{
	printf("Exercicio 3, Lab 5 de CHAD. Efectua a transposta duma matriz.\n");
	int *d_matrix, *d_out, *h_matrix, error, M;
	//Start simple. N = M
	for(int N = 64; N <= 512; N = N * 2){
		M = N;
		printf("Transposta duma matriz %i * %i.\n", N, M);
		//alocar memoria para input do device
		if(hipMalloc(&d_matrix, sizeof(int) * N * M) != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		//alocar memoria para output do device
		if(hipMalloc(&d_out, sizeof(int) * N * M) != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		//alocar memoria para matriz no host
		h_matrix = (int *) malloc(N * M * sizeof(int));
			//inicializar matriz
		for(int i = 0; i < N * M; i++)
			h_matrix[i] = i;

		//dimensionar grid e block sizes
		dim3 BlockSize(32, 32, 1);
		dim3 GridSize(N / 32 + 1, M / 32 + 1, 1);
		//transferir matriz para device.
		if((error = hipMemcpy(d_matrix, h_matrix, N * M * sizeof(int), hipMemcpyHostToDevice)))
			fprintf(stderr, "Erro a transferir matriz para device. Error code: %i.\n", error);

		transposta<<<GridSize, BlockSize>>>(d_matrix, d_out, N);
		if((error = hipMemcpy(h_matrix, d_out, N * M * sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess)
			fprintf(stderr, "Erro a transferir matriz do device para host. Error code: %i.\n", error);
		//imprimir uma sub matriz 5*5
		for(int i = 0; i < 5; i++){
			for(int j = 0; j < 5; j++)
				printf(" %i ", h_matrix[i * N + j]);
			printf("\n");
		}
		if((error = hipFree(d_matrix)) != hipSuccess)
			printf("Erro a libertar memoria no device. Error code: %i.\n", error);
		if((error = hipFree(d_out)) != hipSuccess)
			printf("Erro a libertar memoria no device. Error code: %i.\n", error);
		free(h_matrix);







	}
	return 0;
}