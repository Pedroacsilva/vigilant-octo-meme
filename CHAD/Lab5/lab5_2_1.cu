//nvcc -o lab5_2_1 lab5_2_1.cu
/*Author:
Pedro Silva
*/
/*2. Implemente um programa em CUDA que calcule a soma de todos os elementos de um vetor de
tamanho N. Teste para vários valores de N.*/
/*2.1. Implemente uma versão simples (sem recorrer a optimizações).*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void vectorsum2_1(int * d_buffer, int N){
	//THREAD ID
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	/*Temos N elementos no h_buffer. Vamos partir a soma de todos os elementos como a soma de um elemento com o elemento 16 indices a direita
	Repetir até so termos um elemento (a cada iteração, temos metade dos elementos).*/
	int num_of_threads = N;
	int distance = N / 2;			//Distancia entre elementos a somar
	int primeiro, segundo;
	//Este ciclo executa enquanto tivermos mais que uma thread e so se a thread ainda estiver no "scope" da soma.
	while(num_of_threads > 1 && index < N/2){
		primeiro = index;
		segundo = primeiro + distance;			//na primeira iteracao: 1a thread, index 1, 2a thread, index 3, 3a thread, index 5
		d_buffer[primeiro] = d_buffer[primeiro] + d_buffer[segundo];
		//passou uma iteracao: duplicar a distancia entre elementos a somar e dividir por 2 o numero de threads activas
		distance = distance / 2;
		num_of_threads = num_of_threads / 2;
		//garantir que todas as threads fizeram a sua soma
		__syncthreads();
	}
}		

int main(){
	printf("Exercicio 2, Lab 5 de CHAD. Soma de todos os elementos de um h_buffer de tamanho N.\nN comeca a 8 (2^3)e duplica até 4096 (2^10).\n");
	int *d_buffer, *result, *h_buffer;
	int error;
	struct timespec start, end;
	double startTime, endTime;

	for( int N = 256; N <= 4096; N = N*2){
		printf("--------------------------------------------------------------------------\n");
		printf("Soma de um vector com %i elementos.\n", N);
		clock_gettime(CLOCK_MONOTONIC, &start);
		//alocar memoria no device
		if(hipMalloc(&d_buffer, sizeof(int) * N) != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		//alocar memoria no host para h_buffer
		h_buffer = (int*) malloc(N * sizeof(int));
		for(int i = 0; i < N; i++)
			h_buffer[i] = 1;
		//alocar memoria no host para receber o resultado de cada bloco
		result = (int*) malloc(N * sizeof(int));
		//Transferir dados do device para host (vector a somar)
		if((error = hipMemcpy(d_buffer, h_buffer, N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess)
			fprintf(stderr, "Erro a transferir vector para GPU, de dimensao %i. Error = %i.\n", N, error);
		//Inicializar block e grid size
		dim3 BlockSize(32, 1, 1);		//Comecar simples: Blocos de tamanho fixo
		dim3 GridSize(N/32 + 1, 1, 1);	
		vectorsum2_1<<<GridSize, BlockSize>>>(d_buffer, N);
		//Vamos buscar o resultado da soma ao primeiro elemento do d_buffer
		hipMemcpy(result, d_buffer, sizeof(int), hipMemcpyDeviceToHost);
		printf("Resultado da soma de um vector de %i elementos: %i.\n", N, *result);
		if(hipFree(d_buffer) != hipSuccess)
			printf("Erro a libertar memoria no device.\n");
		clock_gettime(CLOCK_MONOTONIC, &end);
		startTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
		endTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
		printf("Tempo de execução do GPU kernel: %fms.\n", endTime - startTime);
		if(hipFree(d_buffer) != hipSuccess)
			printf("Erro a libertar memoria no device para vector.\n");
		free(h_buffer);
		free(result);
	}

	
	return 0;
}