//nvcc -o lab5_2_2 lab5_2_2.cu
/*Author:
Pedro Silva
*/
/*2. Implemente um programa em CUDA que calcule a soma de todos os elementos de um vetor de
tamanho N. Teste para vários valores de N.*/
/*2.2. Implemente uma nova versão otimizada baseada em memória partilhada.*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void vectorsum2_2(int * d_buffer, int N){
	//THREAD ID
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	/*Temos N elementos no h_buffer. Vamos partir a soma de todos os elementos como a soma de um elemento com o elemento a sua direita
	Repetir até so termos um elemento (a cada iteração, temos metade dos elementos).*/
	//Comecamos por meter tudo em shared memory
	__shared__ int shared_mem[32];		//32 = Block size
	if(index < N){
		shared_mem[threadIdx.x] = d_buffer[index];
		__syncthreads();
	}
	/*int threads_per_block = 16;
	int distance = 16;
	int primeiro, segundo;*/


	/*//Este ciclo executa enquanto tivermos mais que uma thread e so se a thread ainda estiver no "scope" da soma.
	while(threads_per_block > 1 && threadIdx.x < threads_per_block && index < N / 2){
		//Somar par de elementos  em shared memory
		primeiro = threadIdx.x;
		segundo = threadIdx.x + distance;
		shared_mem[primeiro] = shared_mem[primeiro] + shared_mem[segundo];
		threads_per_block = threads_per_block / 2;
		distance = distance / 2;
		//garantir que todas as threads fizeram a sua soma
		__syncthreads();
	}*/

	//Estou a ter problemas em fazer a reducao, pelo que vou fazer algo mais simples. A thread 0 de cada bloco faz a soma de 32 elementos
		if(threadIdx.x == 0){
			for(int i = 1; i < blockDim.x && (index + i) < N; i++){
				shared_mem[0] += shared_mem[i];
		//		printf("shared_mem[0] = %i. i = %i. .blockIdx.x = %i.\n", shared_mem[0], i, blockIdx.x);
			}
		}

	//A primeira thread de cada bloco deve agora meter o resultado da soma do seu bloco no device buffer de acordo com o indice do seu bloco

		if(threadIdx.x == 0){
			d_buffer[blockIdx.x] = shared_mem[0];
		//printf("A thread do bloco %i escreveu %i.\n", blockIdx.x, shared_mem[0]);
		}
	}		

	int main(){
		printf("Exercicio 2, Lab 5 de CHAD. Soma de todos os elementos de um vector de tamanho N.\nN comeca a 8 (2^3)e duplica até 4096 (2^10).\n");
		int *d_buffer, *result, *h_buffer;
		int error;
		struct timespec start, end;
		double startTime, endTime;

		for( int N = 256; N <= 4096; N = N*2){
			printf("--------------------------------------------------------------------------\n");
			printf("Soma de um vector com %i elementos.\n", N);
			clock_gettime(CLOCK_MONOTONIC, &start);
		//alocar memoria no device
			if(hipMalloc(&d_buffer, sizeof(int) * N) != hipSuccess){
				fprintf(stderr, "Error allocating memory on device.\n");
				return(-1);
			}
		//alocar memoria no host para h_buffer
			h_buffer = (int*) malloc(N * sizeof(int));
			for(int i = 0; i < N; i++)
				h_buffer[i] = 1;
		//alocar memoria no host para receber o resultado de cada bloco
			result = (int*) malloc(N * sizeof(int));
		//Transferir dados do host para device (vector a somar)
			if((error = hipMemcpy(d_buffer, h_buffer, N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess)
				fprintf(stderr, "Erro a transferir vector para GPU, de dimensao %i. Error = %i.\n", N, error);
		//Inicializar block e grid size
		dim3 BlockSize(32, 1, 1);		//Comecar simples: Blocos de tamanho fixo
		dim3 GridSize(N/32 + 1, 1, 1);
		printf("Gridsize: (%i, %i, %i).\n",GridSize.x, GridSize.y, GridSize.z);	
		vectorsum2_2<<<GridSize, BlockSize>>>(d_buffer, N);
		//Vamos buscar o resultado da soma ao primeiro elemento do d_buffe
		if((error = hipMemcpy(result, d_buffer, N * sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess)
			fprintf(stderr, "Erro a transferir vector do device para host. Error code: %i.\n", error);
		if((error = hipFree(d_buffer)) != hipSuccess)
			printf("Erro a libertar memoria no device. Error code: %i.\n", error);
		//Temos agora de somar os resultados de cada bloco
		for(int i = 1; i < N / 32 + 1; i ++)
			result[0] += result[i];
		printf("Resultado: %i.\n", result[0]);
		clock_gettime(CLOCK_MONOTONIC, &end);
		startTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
		endTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
		printf("Tempo de execução do GPU kernel: %fms.\n", endTime - startTime);
		if((error = hipFree(d_buffer)) != hipSuccess)
			printf("Erro a libertar memoria no device para vector. Error code: %i.\n", error);
		free(h_buffer);
		free(result);
	}

	
	return 0;
}