//nvcc -o lab5_5_1 lab5_5_1.cu
//Author: Pedro Silva
/*5. Desenvolva um programa em CUDA que implemente a multiplicação de matrizes
5.1. Implemente uma versão simples (sem optimizações) e compare com a versão sequencial.*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixMul(int * d_A, int * d_B, int* d_C, int N, int M){

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	//garantir que estamos no scope do problema
	if(row < M && col < N){
		int index = row * N + col;
		for(int i = 0; i < N; i++){
			d_C[index] += d_A[index] * d_B[index];
		}
	}

}



int main(int argc, char const *argv[])
{
	printf("Exercicio 5.1, Lab 5 de CHAD. Multiplicacao de matrizes (simples) com CUDA.\n");
	int * h_A, * h_B, *d_A, *d_B, *d_C;
	int N = 128;			//Trabalhar com matrizes 128*128
	int M = N;
	//Alocar memoria no host para matriz
	h_A = (int*) malloc(N * N * sizeof(int));
	h_B = (int*) malloc(N * N * sizeof(int));
	//inicializar matrizes A e B
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			h_A[i + j * N] = i;
			h_B[i + j * N] = j;
		}
	}
	//alocar memoria para device
	if(hipMalloc(&d_A, N * M * sizeof(int)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para matriz A.\n");
		return(-1);
	}
	if(hipMalloc(&d_B, N * M * sizeof(int)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para matriz B.\n");
		return(-1);
	}
	if(hipMalloc(&d_C, N * M * sizeof(int)) != hipSuccess){
		fprintf(stderr, "Erro a alocar memória no device para matriz C.\n");
		return(-1);
	}
	//transferir dados de host para device
	if(hipMemcpy(d_A, h_A, N * M * sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		fprintf(stderr, "Erro a transferir matriz A de host para device.\n");
	}
	if(hipMemcpy(d_B, h_B, N * M * sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		fprintf(stderr, "Erro a transferir matriz B de host para device.\n");
	}
	//definir dimensao de block e grid
	dim3 BlockSize(32,32,1);
	dim3 GridSize(N / 32 + 1, M / 32 + 1, 1);
	//lancar GPU kernel
	MatrixMul<<<GridSize, BlockSize>>>(d_A, d_B, d_C, N, M);
	//Transferir matriz de resultados para host
	if(hipMemcpy(h_A, d_C, N * M * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
		fprintf(stderr, "Erro a transferir matriz C de device para host.\n");
	}
	printf("C[%i][%i]: %i.\n", N/2, M/2, h_A[N/2 + M/2 * N]);

	return 0;
}






