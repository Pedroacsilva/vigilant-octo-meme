#include "hip/hip_runtime.h"
//nvcc -o lab5_22 lab5_22.cu
/*Author:Pedro Silva*/
/*2. Implemente um programa em CUDA que calcule a soma de todos os elementos de um vetor de
tamanho N. Teste para vários valores de N.*/
//2.2. Implemente uma nova versão otimizada baseada em memória partilhada.
//Assume each block has 64 threads (cheating but idc)
#define BLOCK_SIZE 64
__global__ void vectorsum2_2(int * device_buffer, int N){
	//THREAD ID
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//Declarar shared memory, BLOCK_SIZE integers
	__shared__ int elements[BLOCK_SIZE];
	/*We have N elements to sum. We'll break the sum of all elements as sums of one element + the element half a block away. For each iteration
	of the sum, halve the number of threads per block. Repeat until we have one thread per block.*/
	/*Each thread loads one element from global to shared memory.*/
	if(index <= N)
		elements[index] = device_buffer[index] ;
	//syncthreads to make sure all elements are loaded into shared memory
	__syncthreads();
	/*For N elements to sum, N/2 threads sum 2 elements together*/
	int threads_per_block = BLOCK_SIZE/2;
	/*Loop where each threads sums 2 elements in shared memory and stores it in shared mem. Halve the number of threads per block
	each iteration, until we have one thread (WARNING: Need to define workflow when number of threads is odd.)*/
	while(threads_per_block > 1 && threadIdx.x <= threads_per_block){
		elements[threadIdx.x] = elements[threadIdx.x] + elements[threadIdx.x + threads_per_block];
		threads_per_block /= 2;
		__syncthreads();	//Make sure each thread has finished computation before doing next iteration
	}
	/*One thread per block, load sum from shared memory to global device memory. Host will then load  and sum those values*/
	if(threadIdx.x == 0)
		device_buffer[index] = elements[0];
}		

int main(){
	//32768 = 2¹⁵;
	for(int N = BLOCK_SIZE; N <= BLOCK_SIZE * 8; N = N*2){
		int *device_buffer = NULL;
		int err = hipMalloc(&device_buffer, sizeof(int) * N);
		if(err != hipSuccess){
			fprintf(stderr, "Error allocating memory on device.\n");
			return(-1);
		}
		vectorsum2_2<<< N/BLOCK_SIZE + 1, BLOCK_SIZE >>>(device_buffer, N);
		int results[N / BLOCK_SIZE];
		//Fetch sum results from device to host
		for(int i = 0; i < N/BLOCK_SIZE; i++)
			hipMemcpy(results[i], device_buffer[i * BLOCK_SIZE], sizeof(int));
		printf("Sum of a vector of %i elements: %i.\n", N, result);
		//FREE DEVICE MEMORY
		hipFree(device_buffer);
	}
	
	return 0;
}


