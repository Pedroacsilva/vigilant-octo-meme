#include "hip/hip_runtime.h"
//nvcc -o lab5_1_gpu lab5_1_gpu.cu
//Implementacao em GPU para a conversao em tons de cinza duma imagem RGB
//Autor: Pedro Silva

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
/*stb -> biblioteca para abrir e ler imagens: https://github.com/nothings/stb*/

#define CHANNELS 4		//(RGBA)

// Device code
// We have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyScaleConvertion(unsigned char* grayImage, unsigned char
	* rgbImage, int width, int height)
{
	int Col = threadIdx.x + (blockIdx.x * blockDim.x);
	int Row = threadIdx.y + (blockIdx.y * blockDim.y);
	if (Col < width && Row < height)
	{
		// get 1D coordinate for the grayscale image
		int grayOffset = Row * width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns of the gray scale image
		int rgbOffset = grayOffset * CHANNELS;
		unsigned char r = rgbImage[rgbOffset];
		// red value for pixel
		unsigned char g = rgbImage[rgbOffset + 1];
		// green value for pixel
		unsigned char b = rgbImage[rgbOffset + 2];
		// blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		grayImage[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
	}
}


int main(int argc, char const *argv[])
{
	printf("Exercicio 1, Lab 5 de CHAD. Leitura e grayscaling de imagens com varias dimensoes (GPU).\n");
	int N, M, channels;
	unsigned char * h_i_rgb, * h_i_gs;
	unsigned char * d_i_gs, * d_i_rgb;
	struct timespec start, end;
	double startTime, endTime;

	//Vamos testar o device code com imagens de dimensoes: 255x255, 800x600, 1920x1080, 3840x2160, 7680x4320, portanto vamos iterar pelo codigo 5 vezes.

	for(int i = 0; i < 5; i++){
		printf("%i-ésima iteração.\n", i);
		//Definir as dimensoes da imagem, N e M, de acordo com a iteracao, i, do ciclo e load da imagem
		//unsigned char *data = stbi_load(filename, &x, &y, &n, 0);
		// Standard parameters:
//    int *x                 -- outputs image width in pixels
//    int *y                 -- outputs image height in pixels
//    int *channels_in_file  -- outputs # of image components in image file
//    int desired_channels   -- if non-zero, # of image components requested in result
		//int i = 1;
		switch(i){
			case 0:
			h_i_rgb = stbi_load("image_255_255.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 1:
			h_i_rgb = stbi_load("image_800_600.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 2:
			h_i_rgb = stbi_load("image_1920_1080.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 3:
			h_i_rgb = stbi_load("image_3840_2160.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			case 4:
			h_i_rgb = stbi_load("image_7680_4320.png", &N, &M, &channels, 0);
			if(h_i_rgb == NULL)
				fprintf(stderr, "Erro a carregar imagem na iteração %i.\n", i);
			printf("Dimensoes da imagem: %i x %i.\tNumero de canais: %i.\n", N, M, channels);
			break;
			default:
			fprintf(stderr, "Erro no switch para inicializar dimensoes da imagem.\n");
			return(-1);
		}
		//Inicializar block e grid size
		dim3 BlockSize(32, 32, 1);		//Comecar simples: Blocos de tamanho fixo
		dim3 GridSize(N / 32 + 1, M / 32 + 1, 1);	

		//Alocar memoria no device para imagem em RGB (entrada) e imagem em grayscale (saida)
		if(hipMalloc(&d_i_rgb, N * M * channels * sizeof(unsigned char)) != hipSuccess){
			fprintf(stderr, "Erro a alocar memória no device para imagem RGB na iteração %i.\n", i);
			return(-1);
		}
		if(hipMalloc(&d_i_gs, N * M * 1 * sizeof(unsigned char)) != hipSuccess){
			fprintf(stderr, "Erro a alocar memória no device para imagem GS na iteração %i.\n", i);
			return(-1);
		}
		//Transferir dados de device para host (imagem RGB)
		if(hipMemcpy(d_i_rgb, h_i_rgb, N * M * channels * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess){
			fprintf(stderr, "Erro a transferir imagem RGB de host para device.\n");
		}
		//Alocar memoria para receber imagem
		h_i_gs = (unsigned char*)malloc(N * M * 1 * sizeof(unsigned char));		//So temos um canal de cinzentos
		//Lancar kernel do device e cronometrar
		clock_gettime(CLOCK_MONOTONIC, &start);
		colorToGreyScaleConvertion<<<BlockSize, GridSize>>>(d_i_gs, d_i_rgb, N, M);

		//Transferir dados de device->host
		if(hipMemcpy(h_i_gs, d_i_gs, N * M * 1 * sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess){
			fprintf(stderr, "Erro a transferir iamgem GS de device para host.\n");
			return(-1);
		}
		//Gravar imagem em ficheiros
		//int stbi_write_png(char const *filename, int w, int h, int comp, const void *data, int stride_in_bytes);
		switch(i){
			case 0:
			if(stbi_write_png("image_255_255_grey_gpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 1:
			if(stbi_write_png("image_800_600_grey_gpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 2:
			if(stbi_write_png("image_1920_1080_grey_gpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 3:
			if(stbi_write_png("image_3840_2160_grey_gpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			case 4:
			if(stbi_write_png("image_7680_4320_grey_gpu.png", N, M, 1, h_i_gs, N) == 0)
				fprintf(stderr, "Erro a imprimir imagem de %i por %i.\n", N, M);
			break;
			default:
			fprintf(stderr, "Erro no switch de imprimir imagem.\n");
			return(-1);
		}
	/*		switch(i){
				case 0:
				stbi_write_bmp("image_255_255_grey_gpu.bmp", N, M, 1, h_i_gs);
				break;
				case 1:
				stbi_write_bmp("image_800_600_grey_gpu.bmp", N, M, 1, h_i_gs);
				break;
				case 2:
				stbi_write_bmp("image_1920_1080_grey_gpu.bmp", N, M, 1, h_i_gs);
				break;
				case 3:
				stbi_write_bmp("image_3840_2160_grey_gpu.bmp", N, M, 1, h_i_gs);
				break;
				case 4:
				stbi_write_bmp("image_7680_4320_grey_gpu.bmp", N, M, 1, h_i_gs);
				break;
				default:
				fprintf(stderr, "Erro no switch de imprimir imagem.\n");
				return(-1);
			}*/



		//Memory cleanup
				if(hipFree(d_i_rgb) != hipSuccess)
					printf("Erro a libertar memoria no device para imagem RGB.\n");
				if(hipFree(d_i_gs) != hipSuccess)
					printf("Erro a libertar memoria no device para imagem GS.\n");
				stbi_image_free(h_i_rgb);
				free(h_i_gs);
				clock_gettime(CLOCK_MONOTONIC, &end);
				startTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
				endTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
				printf("Tempo de execução do GPU kernel: %fms.\n", endTime - startTime);


			}



			return 0;
		}








